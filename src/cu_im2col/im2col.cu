#include "mex.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "hemi\hemi.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		char err_str[1000];
		sprintf(err_str,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		mexErrMsgTxt(err_str);
		
	}
}

#define HEMI_GRID_STRIDE_LOOP(iter, num) 	for (int iter = hemiGetElementOffset(); \
	iter<num;\
	iter+=hemiGetElementStride())



HEMI_KERNEL(d_im2col)(float* d_image,  int img_c_size, int img_r_size,
					  int thread_num, int total_length, 
					  int ksize_c, int ksize_r, int channels, 
					  int stride_c, int stride_r,		  
					  int padding_c, int padding_r, float* d_output)
{
	int out_c_size, out_r_size, out_row, out_col, ch;
	int col_length = ksize_c*ksize_r*channels;
	int ksize = ksize_c*ksize_r;
	int img_channel_size = img_c_size*img_r_size;
	out_c_size = (img_c_size-ksize_c+2*padding_c)/stride_c+1;
	out_r_size = (img_r_size-ksize_r+2*padding_r)/stride_r+1;
	HEMI_GRID_STRIDE_LOOP(idx, thread_num){

		//transform the image data to columns
		
		int index = idx;
		out_row = index%out_c_size;
		index/=out_c_size;
		out_col = index%out_r_size;
		ch = index/out_r_size;
		int col_base = col_length*(idx%total_length)+ksize*ch;
		int img_base = img_channel_size*ch;

		int ori_c_zero= out_col*stride_c-padding_c;
		int ori_r_zero= out_row*stride_r-padding_r;

			for (int k_c=0; k_c<ksize_r; k_c++)
				for (int k_r=0; k_r<ksize_c; k_r++){
					int ori_c = ori_c_zero+k_c;
					int ori_r = ori_r_zero+k_r;
					d_output[col_base+ k_c*ksize_c+k_r]=(ori_c>=0&&ori_c<img_c_size&&ori_r>=0&&ori_r<img_r_size)?
						d_image[img_base+ori_c*img_c_size+ori_r]
						//ch*ksize+ k_c*ksize_c+k_r
					:0;
					//d_output[idx] = col_base;
				}

	}
}

#define THREAD_PER_BLOCK 256

#define IMG_OUT plhs[0]

#define IMG_IN prhs[0]
#define KSIZE_IN prhs[1]
#define STRIDE_IN prhs[2]
#define PADDING prhs[3]

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	//All code and internal function calls go in here!
	if(nrhs<2)
		mexErrMsgTxt("Not enough inputs");

	double* stride_in;
	size_t stride_c, stride_r;
	if(nrhs>=3){

		stride_in = (double*) mxGetData(STRIDE_IN);
		stride_c = (size_t)stride_in[0];
		stride_r = (size_t)stride_in[1];
	}
	else{
		stride_c = 1;
		stride_r = 1;
	}



	int padding=0;
	if(nrhs==4)
		padding = mxGetScalar(PADDING);

	float *img_in;
	float *img_out;
	img_in = (float*)mxGetData(IMG_IN);


	double* filter_size;
	filter_size = (double*) mxGetData(KSIZE_IN);
	size_t k_height, k_width;
	k_height = (size_t)filter_size[0];
	k_width = (size_t)filter_size[1];

	




	size_t img_height =mxGetDimensions(IMG_IN)[0];
	size_t img_width = mxGetDimensions(IMG_IN)[1];
	size_t img_channel = mxGetDimensions(IMG_IN)[2];

	//mexPrintf("height %d width %d channel %d\n", img_height, img_width, img_channel);
	//mexPrintf("filter height %d filter width %d \n", k_height, k_width);
	

	size_t total_size = ((img_height+2*padding-k_height)/stride_c+1)*((img_width+2*padding-k_width)/stride_r+1);
	size_t col_size = k_height*k_width*img_channel;
	int thread_num  = total_size*img_channel;

	//mexPrintf("Total %d columns \n", total_size);
	
	IMG_OUT = mxCreateNumericMatrix(col_size, total_size, mxSINGLE_CLASS, mxREAL);
	img_out = (float*)mxGetData(IMG_OUT);


	/* Start CUDA PROCESSING*/
	float *d_img;
	float *d_col;
	size_t n_pixels = img_height*img_width*img_channel;
	gpuErrchk(hipMalloc(&d_img, n_pixels*sizeof(float)));
	gpuErrchk(hipMalloc(&d_col, col_size*total_size*sizeof(float)));


	/* Copy data to GPU mem */
	//copy data
	gpuErrchk(hipMemcpy(d_img, img_in, n_pixels*sizeof(float), hipMemcpyHostToDevice));

	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

	
	/* launch kernel */
	HEMI_KERNEL_LAUNCH(d_im2col, 32*numSMs, THREAD_PER_BLOCK, 0, 0, 
		d_img, img_height, img_width, 
		thread_num, total_size,
		k_height, k_width, img_channel,
		stride_c,stride_r,
		padding, padding, 
		d_col);
	gpuErrchk(hipDeviceSynchronize());
	/*copy result back to cpu mem */
	gpuErrchk(hipMemcpy(img_out, d_col, col_size*total_size*sizeof(float), hipMemcpyDeviceToHost));



	gpuErrchk(hipFree(d_img));
	gpuErrchk(hipFree(d_col));
	return;
}
